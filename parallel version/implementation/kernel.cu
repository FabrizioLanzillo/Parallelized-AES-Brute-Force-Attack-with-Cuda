#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

#include <stdio.h>
#include <fstream>
#include <cstring>
#include <string.h>
#include <sstream>  
#include <hip/hip_runtime.h>
#include <stdint.h>

using namespace std;

//Encrypting/Decrypting Parameters definition
#define AES_KEYLENGTH 32
#define IV_KEYLENGTH 16
#define SALT_KEYLENGTH 8
#define DEBUG true
#define BLOCK_SIZE 16
#define PLAINTEXT_LENGHT 445
#define AES_keyExpSize 240


// The number of columns comprising a state in AES. This is a constant in AES. Value=4
#define Nb 4
#define Nk 8
#define Nr 14
#define MULTIPLY_AS_A_FUNCTION 0


//Brute Force configuration
#define BASE_NUMBER 2

struct AES_ctx
{
	uint8_t RoundKey[AES_keyExpSize];
	uint8_t Iv[BLOCK_SIZE];

};

//              PARAMETERS
//  Key generated from openssl enc -aes-256-cbc -key_aes secret -P -md sha1
//  salt = B51DE47CC865460E
//  key = 85926BE3DA736F475493C49276ED17D418A55A2CFD077D1215ED251C4A57D8EC
//  85 92 6B E3 DA 73 6F 47 54 93 C4 92 76 ED 17 D4 18 A5 5A 2C FD 07 7D 12 15 ED 25 1C 4A 57 D8 EC  
//  iv = D8596B739EFAC0460E861F9B7790F996
//  iv =D8 59 6B 73 9E FA C0 46 0E 86 1F 9B 77 90 F9 96

//Key in HEX format as global parameters
//static const int key_size = 32;
//const int num_bits_to_hack = 12;
const string plaintext_file = "./../../files/text_files/plaintext.txt";
const string ciphertext_file = "./../../files/text_files/ciphertext.txt";
const string key_aes_hex_file = "./../../files/secret_files/key_aes_hex.txt";
const string key_aes_file = "./../../files/secret_files/key_aes.txt";
//const string key_wrong_file = "key_wrong.txt";
//const string key_wrong_file_hex = "key_wrong_hex.txt";
const string iv_file_hex = "./../../files/secret_files/iv_hex.txt";
const string iv_file = "./../../files/secret_files/iv.txt";
const string salt_file_hex = "./../../files/secret_files/salt_hex.txt";
const string salt_file = "./../../files/secret_files/salt.txt";

/*****************************************************************************/
/* Private variables:                                                        */
/*****************************************************************************/
// state - array holding the intermediate results during decryption.
typedef uint8_t state_t[4][4];

/* ***************************************************************************************************/
/* ******************************************* CONSTANTS *********************************************/
/* ***************************************************************************************************/

// The lookup-tables are marked const so they can be placed in read-only storage instead of RAM
// The numbers below can be computed dynamically trading ROM for RAM - 
// This can be useful in (embedded) bootloader applications, where ROM is often limited.
const uint8_t sbox[256] = {
	//0     1    2      3     4    5     6     7      8    9     A      B    C     D     E     F
	0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
	0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
	0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
	0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
	0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
	0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
	0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
	0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
	0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
	0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
	0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
	0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
	0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
	0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
	0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
	0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };


// The round constant word array, Rcon[i], contains the values given by 
// x to the power (i-1) being powers of x (x is denoted as {02}) in the field GF(2^8)
const uint8_t Rcon[11] = {
  0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };

__device__ const uint8_t d_rsbox[256] = {
  0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
  0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
  0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
  0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
  0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
  0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
  0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
  0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
  0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
  0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
  0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
  0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
  0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
  0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
  0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
  0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d };

/* *************************************************************************************************/
/* ******************************************* DEC_FUN *********************************************/
/* *************************************************************************************************/

__device__ void XorWithIv(uint8_t* buf, const uint8_t* Iv)
{
	uint8_t i;
	for (i = 0; i < BLOCK_SIZE; ++i) // The block in AES is always 128bit no matter the key size
	{
		buf[i] ^= Iv[i];
	}
}

/** Extracts a specific value from the SBOX
* num: cell to extract
*/
uint8_t getSBoxValue(uint8_t num)
{
	return sbox[num];
}

/* This function produces Nb(Nr + 1) round keys.The round keys are used in each round to decrypt the states.
* RoundKey: rounded key
* Key: original key
*/
void KeyExpansion(uint8_t* RoundKey, const uint8_t* Key)
{
	unsigned i, j, k;
	uint8_t tempa[4]; // Used for the column/row operations

	// The first round key is the key itself.
	for (i = 0; i < Nk; ++i)
	{
		RoundKey[(i * 4) + 0] = Key[(i * 4) + 0];
		RoundKey[(i * 4) + 1] = Key[(i * 4) + 1];
		RoundKey[(i * 4) + 2] = Key[(i * 4) + 2];
		RoundKey[(i * 4) + 3] = Key[(i * 4) + 3];
	}

	// All other round keys are found from the previous round keys.
	for (i = Nk; i < Nb * (Nr + 1); ++i)
	{
		{
			k = (i - 1) * 4;
			tempa[0] = RoundKey[k + 0];
			tempa[1] = RoundKey[k + 1];
			tempa[2] = RoundKey[k + 2];
			tempa[3] = RoundKey[k + 3];

		}

		if (i % Nk == 0)
		{
			// This function shifts the 4 bytes in a word to the left once.
			// [a0,a1,a2,a3] becomes [a1,a2,a3,a0]

			// Function RotWord()
			{
				const uint8_t u8tmp = tempa[0];
				tempa[0] = tempa[1];
				tempa[1] = tempa[2];
				tempa[2] = tempa[3];
				tempa[3] = u8tmp;
			}

			// SubWord() is a function that takes a four-byte input word and 
			// applies the S-box to each of the four bytes to produce an output word.

			// Function Subword()
			{
				tempa[0] = getSBoxValue(tempa[0]);
				tempa[1] = getSBoxValue(tempa[1]);
				tempa[2] = getSBoxValue(tempa[2]);
				tempa[3] = getSBoxValue(tempa[3]);
			}

			tempa[0] = tempa[0] ^ Rcon[i / Nk];
		}
		if (i % Nk == 4)
		{
			// Function Subword()
			{
				tempa[0] = getSBoxValue(tempa[0]);
				tempa[1] = getSBoxValue(tempa[1]);
				tempa[2] = getSBoxValue(tempa[2]);
				tempa[3] = getSBoxValue(tempa[3]);
			}
		}
		j = i * 4; k = (i - Nk) * 4;
		RoundKey[j + 0] = RoundKey[k + 0] ^ tempa[0];
		RoundKey[j + 1] = RoundKey[k + 1] ^ tempa[1];
		RoundKey[j + 2] = RoundKey[k + 2] ^ tempa[2];
		RoundKey[j + 3] = RoundKey[k + 3] ^ tempa[3];
	}
}

void AES_init_ctx_iv(struct AES_ctx* ctx, const uint8_t* key, const uint8_t* iv)
{
	KeyExpansion(ctx->RoundKey, key);
	memcpy(ctx->Iv, iv, BLOCK_SIZE);
}

/* This function adds the round key to state. The round key is added to the state by an XOR function.
* round: state variable containing the round number
* state: state variable containing the current state of pt to ct conversion for this round
* RoundKey: contains the key rounded for the current round
*/
__device__ void AddRoundKey(uint8_t round, state_t* state, const uint8_t* RoundKey)
{
	uint8_t i, j;
	for (i = 0; i < 4; ++i)
	{
		for (j = 0; j < 4; ++j)
		{
			(*state)[i][j] ^= RoundKey[(round * Nb * 4) + (i * Nb) + j];
		}
	}
}

__device__ uint8_t xtime(uint8_t x)
{
	return ((x << 1) ^ (((x >> 7) & 1) * 0x1b));
}

__device__ uint8_t Multiply(uint8_t x, uint8_t y)
{
	return (((y & 1) * x) ^
		((y >> 1 & 1) * xtime(x)) ^
		((y >> 2 & 1) * xtime(xtime(x))) ^
		((y >> 3 & 1) * xtime(xtime(xtime(x)))) ^
		((y >> 4 & 1) * xtime(xtime(xtime(xtime(x)))))); /* this last call to xtime() can be omitted */
}

__device__ uint8_t getSBoxInvert(uint8_t num)
{
	return d_rsbox[num];
}

/** MixColumns function mixes the columns of the state matrix. 
*  state: state variable containing the current state of pt to ct conversion for this round
*/ 
__device__ void InvMixColumns(state_t* state)
{
	int i;
	uint8_t a, b, c, d;
	for (i = 0; i < 4; ++i)
	{
		a = (*state)[i][0];
		b = (*state)[i][1];
		c = (*state)[i][2];
		d = (*state)[i][3];

		(*state)[i][0] = Multiply(a, 0x0e) ^ Multiply(b, 0x0b) ^ Multiply(c, 0x0d) ^ Multiply(d, 0x09);
		(*state)[i][1] = Multiply(a, 0x09) ^ Multiply(b, 0x0e) ^ Multiply(c, 0x0b) ^ Multiply(d, 0x0d);
		(*state)[i][2] = Multiply(a, 0x0d) ^ Multiply(b, 0x09) ^ Multiply(c, 0x0e) ^ Multiply(d, 0x0b);
		(*state)[i][3] = Multiply(a, 0x0b) ^ Multiply(b, 0x0d) ^ Multiply(c, 0x09) ^ Multiply(d, 0x0e);
	}
}

/** The SubBytes Function Substitutes the values in the state matrix with values in an S-box.
* state:  state variable containing the current state of pt to ct conversion for this round
*/
__device__ void InvSubBytes(state_t* state)
{
	uint8_t i, j;
	for (i = 0; i < 4; ++i)
	{
		for (j = 0; j < 4; ++j)
		{
			(*state)[j][i] = getSBoxInvert((*state)[j][i]);
		}
	}
}

/* Reverse of the shift row operation
* state: state variable containing the current state of pt to ct conversion for this round
*/
__device__ void InvShiftRows(state_t* state)
{
	uint8_t temp;

	// Rotate first row 1 columns to right  
	temp = (*state)[3][1];
	(*state)[3][1] = (*state)[2][1];
	(*state)[2][1] = (*state)[1][1];
	(*state)[1][1] = (*state)[0][1];
	(*state)[0][1] = temp;

	// Rotate second row 2 columns to right 
	temp = (*state)[0][2];
	(*state)[0][2] = (*state)[2][2];
	(*state)[2][2] = temp;

	temp = (*state)[1][2];
	(*state)[1][2] = (*state)[3][2];
	(*state)[3][2] = temp;

	// Rotate third row 3 columns to right
	temp = (*state)[0][3];
	(*state)[0][3] = (*state)[1][3];
	(*state)[1][3] = (*state)[2][3];
	(*state)[2][3] = (*state)[3][3];
	(*state)[3][3] = temp;
}

/* implementation of the decryption operation
* state: state variable containing the current state of pt to ct conversion for this round
* RoundKey: rounded key for the round
*/
__device__ void InvCipher(state_t* state, const uint8_t* RoundKey)
{
	uint8_t round = 0;

	// Add the First round key to the state before starting the rounds.
	AddRoundKey(Nr, state, RoundKey);

	// There will be Nr rounds.
	// The first Nr-1 rounds are identical.
	// These Nr rounds are executed in the loop below.
	// Last one without InvMixColumn()
	for (round = (Nr - 1); ; --round)
	{
		InvShiftRows(state);
		InvSubBytes(state);
		AddRoundKey(round, state, RoundKey);
		if (round == 0) {
			break;
		}
		InvMixColumns(state);
	}

}

__global__ void AES_CBC_decrypt_buffer(struct AES_ctx* ctx, unsigned char* ct,unsigned char* pt,size_t length)
{
	size_t i;
	uint8_t storeNextIv[BLOCK_SIZE];
	printf("CIAO\n");
	for (i = 0; i < length; i += BLOCK_SIZE)
	{
		memcpy(storeNextIv, ct, BLOCK_SIZE);
		InvCipher((state_t*)ct, ctx->RoundKey);
		XorWithIv(ct, ctx->Iv);
		memcpy(ctx->Iv, storeNextIv, BLOCK_SIZE);
		ct += BLOCK_SIZE;
		printf("CIAO\n");
	}

	printf("%s\n", ct);
}

/* *************************************************************************************************/
/* ******************************************* UTILITY *********************************************/
/* *************************************************************************************************/

/** Perfrom a read from a file
 * file: name of the file to read
 */
__host__ string read_data_from_file(string file){

	fstream getFile;
	string str;
	string file_contents;
	getFile.open(file,ios::in | ios::binary);

	while (getline(getFile, str)){
		file_contents += str;
		file_contents.push_back('\n');
	} 

	file_contents.pop_back();
	
	getFile.close();
	
	return file_contents;
}

/** Function that perform a conversion from Hexadecimal number into their ASCII representation
 * hex: string that contains the Hexadecimal rapresentation of the text 
 */
__host__ string hexToASCII(string hex){

    // initialize the ASCII code string as empty.
    string ascii = "";
    for (size_t i = 0; i < hex.length(); i += 2)
    {
        // extract two characters from hex string
        string part = hex.substr(i, 2);
 
        // change it into base 16 and
        // typecast as the character
        char ch = stoul(part, nullptr, 16);
        // add this char to final ASCII string
        ascii += ch;
    }
    return ascii;
}

/** Perform a convertion of the key from exadecimal to ASCII and save it on another file
 * file_to_read: file on which we read the exadecimal format key
 * file_to_save: file on which we save the converted key
 */
__host__ void convert_key(string file_to_read, string file_to_save){
	string str = read_data_from_file(file_to_read);
	ofstream SaveFile(file_to_save, ios::out | ios::binary);
	SaveFile << hexToASCII(str);
	SaveFile.close();

}


int main (int argc, char **argv){
	
	/* ------------------------------------- GET KEY -------------------------------------------------------- */
	printf("------------------------------------- GET KEY --------------------------------------------------------\n");
	
	convert_key(iv_file_hex, iv_file);
	convert_key(key_aes_hex_file, key_aes_file);
	unsigned char* iv_aes = (unsigned char*)malloc(IV_KEYLENGTH);
	if(!iv_aes){
		printf ("ERROR: iv space allocation went wrong\n");
		return -1;
	}
	memset(iv_aes, 0, IV_KEYLENGTH);
	strcpy((char*)iv_aes, (char*)read_data_from_file(iv_file).c_str());
	if(DEBUG){
		printf ("IV: %s\n", iv_aes);
	}

	
	unsigned char* key_aes = (unsigned char*)malloc(AES_KEYLENGTH);
	if(!key_aes){
        printf ("ERROR: key space allocation went wrong\n");
		return -1;
	}
	memset(key_aes,0,AES_KEYLENGTH);
	strcpy((char*)key_aes, (char*)read_data_from_file(key_aes_file).c_str());
	if(DEBUG){
        printf ("KEY TO ENCRYPT: %s With length: %lu\n", key_aes, (uint32_t)strlen((char*)key_aes));
	}

    printf("------------------------------------------------------------------------------------------------------\n");
	/* ------------------------------------- GET PT -------------------------------------------------------- */
	printf("------------------------------------- GET PT ---------------------------------------------------------\n");



	//Allocating pt space
	unsigned char* plaintext = (unsigned char*)malloc(PLAINTEXT_LENGHT);
	if(!plaintext){
		printf ("ERROR: plaintext space allocation went wrong\n");
		return -1;
	}
	memset(plaintext,0,PLAINTEXT_LENGHT);
	strcpy((char*)plaintext, (char*)read_data_from_file(plaintext_file).c_str());

	if(DEBUG){
		printf("DEBUG: The Plaintext is: %s\n", plaintext);
	}

	printf("------------------------------------------------------------------------------------------------------\n");
	/* ------------------------------------- GET CT -------------------------------------------------------- */
	printf("------------------------------------- GET CT ---------------------------------------------------------\n");

	//Allocating ct space
	const uint32_t CT_LEN = PLAINTEXT_LENGHT + 3;
	unsigned char* ciphertext = (unsigned char*)malloc(CT_LEN);
	if (!ciphertext) {
		printf("ERROR: CT space allocation went wrong\n");
		return -1;
	}
	memset(ciphertext, 0, CT_LEN);
	strcpy((char*)ciphertext, (char*)read_data_from_file(ciphertext_file).c_str());

	if (DEBUG) {
		printf("DEBUG: The Ciphertext is: %s\n", ciphertext);
	}

	printf("------------------------------------------------------------------------------------------------------\n");
	/* ------------------------------------------ DEC ------------------------------------------------------------ */
	printf("---------------------------------------- DEC ----------------------------------------------------------\n");

	//"d_" variables are the device ones
	struct AES_ctx d_ctx;
	unsigned char* d_key_aes, *d_iv_aes, *d_ciphertext, *d_plaintext;

	//Allocation of variables needed for decryption
	hipError_t rc = hipMalloc((void**)&d_ctx,sizeof(AES_ctx));
	if (rc != hipSuccess) {
		printf("Errore durante allocazione: 1!\n");
		return -1;
	}
	rc = hipMalloc((void**)&d_key_aes, AES_KEYLENGTH);
	if (rc != hipSuccess) {
		printf("Errore durante allocazione: 2!\n");
		return -1;
	}
	rc = hipMalloc((void**)&d_iv_aes, IV_KEYLENGTH);
	if (rc != hipSuccess) {
		printf("Errore durante allocazione: 3!\n");
		return -1;
	}
	rc = hipMalloc((void**)&d_plaintext, 445);
	if (rc != hipSuccess) {
		printf("Errore durante allocazione: 4!\n");
		return -1;
	}
	rc = hipMalloc((void**)&d_ciphertext, 448);
	if (rc != hipSuccess) {
		printf("Errore durante allocazione: 5!\n");
		return -1;
	}
	printf("CIAO\n");
	//Copy the variables value on GPU dynamic memory
	hipMemcpy(d_key_aes, &key_aes, AES_KEYLENGTH, hipMemcpyHostToDevice);
	hipMemcpy(d_iv_aes, &iv_aes, IV_KEYLENGTH, hipMemcpyHostToDevice);
	hipMemcpy(d_plaintext, &plaintext, 445, hipMemcpyHostToDevice);
	hipMemcpy(d_ciphertext, &ciphertext, 448, hipMemcpyHostToDevice);
	printf("CIAO\n");
	//Set the ciphertext context
	AES_init_ctx_iv(&d_ctx, d_key_aes, (uint8_t*)d_iv_aes);
	printf("CIAO\n");

	AES_CBC_decrypt_buffer<<<1,1>>>(&d_ctx, d_ciphertext, d_plaintext, 448);

	return 0;

}