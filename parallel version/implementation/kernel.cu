#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <string>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "header.cuh"

using namespace std;

/******************************************* HOST UTILITY FUNCTION ******************************************/

/**
 * function that read text from file
 * 
 * @param file in input to read
 */
__host__ string read_data_from_file(string file) {

    fstream getFile;
    string str;
    string file_contents;
    getFile.open(file, ios::in | ios::binary);

    while (getline(getFile, str)) {
        file_contents += str;
        file_contents.push_back('\n');
    }

    file_contents.pop_back();

    getFile.close();

    return file_contents;
}

/**
 * function that convert hex characters into a string
 * 
 * @param hex is a string parameter with the hexs character
 */
__host__ string hexToASCII(string hex){

    // initialize the ASCII string
    string ascii = "";
    for (size_t i = 0; i < hex.length(); i += 2){
        
        // two characters from hex string
        string part = hex.substr(i, 2);
        // change into base 16 and cast to char
        char ch = stoul(part, nullptr, 16);
        // add to the ASCII string
        ascii += ch;
    }
    return ascii;
}


/******************************************* PARALLEL DEVICE DECRYPTION ******************************************/

/**
 * function that implement the AES_CBC algorithm and that call the single aes operation
 * 
 * @param state_matrix is the state matrix which element are trasnformed through all the phases
 */
__device__ void single_block_decrypt(uint8_t *state_matrix, uint8_t *iv,const uint8_t* key) {

    AES_round_secret AES_secret;

    //Initialize the secret elements i.e. simmetric key and IV
    initialize_AES_round_secret(&AES_secret, key, iv);
    struct AES_round_secret* rs = &AES_secret;
    // use of AES 256
    decryption_rounds((state_t*)state_matrix, rs->expanded_key);
    // use of the Cipher Block Chaining (CBC)
    xor_with_iv(state_matrix, rs->round_iv); 

    return;
}


/******************************************* PARALLEL DEVICE HACK ******************************************/


/**
 * function that is called from host and executed on the device 
 * 
 * @param device_ciphertext is the ciphertext allocated on the device and that is going to be decrypted
 */
__global__ void kernel_hack(uint8_t* device_ciphertext, uint8_t* device_plaintext, uint8_t* device_cbc_iv, size_t message_num_block, uint8_t* device_key_to_hack, uint8_t* device_return_key){
    
    uint32_t index = threadIdx.x + (blockIdx.x * blockDim.x);

    if (index < message_num_block) {

        // declaration of the data structure to implement the hack
        unsigned char bytes_to_hack[(NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE) + 1];
        uint8_t hacked_key[AES_KEY_BYTES_LENGTH];
        uint8_t state_matrix[AES_BLOCK_LENGTH];
        char ascii_character;
        uint8_t* current_index_to_try = (uint8_t*)&index;
        uint8_t numcycles = NUMBER_BITS_TO_HACK + 1;

        // allocation of the current element for the hacked key
        memcpy(state_matrix, device_ciphertext, AES_BLOCK_LENGTH);
        memcpy(hacked_key, device_key_to_hack, AES_KEY_BYTES_LENGTH); 
        memset(bytes_to_hack,0, (NUMBER_BITS_TO_HACK/NUMBER_BITS_IN_A_BYTE) + 1);

        uint8_t bits_to_maintain = device_key_to_hack[AES_KEY_BYTES_LENGTH - 1 - (NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE)];

        // First copy the bytes that are multiple of 8 bits
        for ( uint32_t j = 0; j <  numcycles; j++ ){
            
            // code that will be executed only if there are remaining bits that are not multiples of 8 bits 
            if( NUMBER_BITS_TO_HACK % NUMBER_BITS_IN_A_BYTE != 0 && j == ( NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE ) ){
                // The addition of unsigned number perform the append correctly until the value inside current_index_to_try[j] 
                // overcome the capacity of the bit to be copied, 
                // but this will never happen since we stop the cycle before it happen
                bytes_to_hack[j] = bits_to_maintain + current_index_to_try[j];
                continue;
            }
            ascii_character = char(index >> (NUMBER_BITS_IN_A_BYTE * j));
            memcpy(&bytes_to_hack[j], &ascii_character, 1);
        }

        // merge of the bits to hack inside the known key
        for (uint32_t j = 0; j < (NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE) + 1; j++) {
            if ( NUMBER_BITS_TO_HACK % NUMBER_BITS_IN_A_BYTE != 0 ) {
                memcpy(&hacked_key[AES_KEY_BYTES_LENGTH - j - 1], &bytes_to_hack[j], 1);
            }
            else if ( j < ( NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE ) ) {
                memcpy(&hacked_key[AES_KEY_BYTES_LENGTH - j - 1], &bytes_to_hack[j], 1);
            }
        }

        __syncthreads();
        
        single_block_decrypt(state_matrix, iv_aes, hacked_key);

        __syncthreads();

        for (uint32_t k = 0; k < AES_BLOCK_LENGTH; k++) {
             
            if ((state_matrix[k] == device_plaintext[k])) {
                if (k == (AES_BLOCK_LENGTH - 1)) {
                    
                    printf("Known key:\t");
                    for (uint32_t k = 0; k < AES_KEY_BYTES_LENGTH; k++){
                        printf("%d|", device_key_to_hack[k]);
                    }
                    printf("\n\n");
                    printf("KEY HACKED!\n\n");
                    printf("Hacked key:\t");
                    for (uint32_t k = 0; k < AES_KEY_BYTES_LENGTH; k++){
                        printf("%d|", hacked_key[k]);
                    }
                    printf("\n");
                    printf("Expected key:\t");
                    for (uint32_t k = 0; k < AES_KEY_BYTES_LENGTH; k++){
                        printf("%d|", key_aes[k]);
                    }
                    printf("\n");

                    memcpy(device_return_key, hacked_key, AES_KEY_BYTES_LENGTH);

                    return;
                }
            }
            else {
                return;
            }
        }

    }

}




int main() {

    /******************************************** SET GPU PROPERTIES **************************************************/


    // inizialize of a struct with all the gpu properties 
    hipDeviceProp_t prop;                   
    // we define the field of the previous struct with the properties of the specified device
    // prop is the struct and the second paramether is the device number
    hipGetDeviceProperties(&prop, 0);  

    /************************************** GET THE PLAINTEXT AND CIPHERTEXT ******************************************/

    // allocate of the plaintext space and read it from file
    unsigned char* plaintext = (unsigned char*)malloc(PLAINTEXT_LENGTH);
	if(!plaintext){
        printf("ERROR: plaintext space allocation went wrong\n");
		return -1;
	}
	memset(plaintext,0,PLAINTEXT_LENGTH);
	strcpy((char*)plaintext, (char*)read_data_from_file(plaintext_file).c_str());

    // allocate of the ciphertext space and read it from file
    unsigned char* ciphertext = (unsigned char*)malloc(CIPHERTEXT_LENGTH);
	if(!ciphertext){
        printf("ERROR: plaintext space allocation went wrong\n");
		return -1;
	}
	memset(ciphertext,0,CIPHERTEXT_LENGTH);

    string file_contents = hexToASCII(read_data_from_file(ciphertext_file));
	// convert to unsigned char
	for(int i=0; i<CIPHERTEXT_LENGTH; i++){
		ciphertext[i] = file_contents[i];
	}

    /************************************* KEY HACK CREATION ********************************************/

    //Creation of the key to hack
    uint8_t key_to_hack[AES_KEY_BYTES_LENGTH];

    //Copy the whole key
    memcpy(&key_to_hack, key_aes_host, AES_KEY_BYTES_LENGTH);

    //Clear the unknown part
    memset(&key_to_hack[AES_KEY_BYTES_LENGTH - (NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE)], 0, NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE);

    //This part must be executed only if there is a part of a byte remaining to be inserted (like last 4 bits in case of 20 bits)
    uint8_t rem_bits = NUMBER_BITS_TO_HACK % NUMBER_BITS_IN_A_BYTE;

    //Use the shift to clean up the part that we don't know of the last byte (like 4 bits in case of 20 bits to discover)
    if (NUMBER_BITS_TO_HACK % NUMBER_BITS_IN_A_BYTE != 0) {
        //With 20 bits -> 2
        key_to_hack[AES_KEY_BYTES_LENGTH - 1 - (NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE)] = key_to_hack[AES_KEY_BYTES_LENGTH - 1 - (NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE)] >> rem_bits;
        key_to_hack[AES_KEY_BYTES_LENGTH - 1 - NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE] = key_to_hack[AES_KEY_BYTES_LENGTH -1  - NUMBER_BITS_TO_HACK / NUMBER_BITS_IN_A_BYTE] << rem_bits;
    }

    /************************************* ALLOCATE AND COPY ON THE DEVICE ********************************************/

    // declaration of the device variable
    uint8_t* device_ciphertext;
    uint8_t* device_plaintext;
    uint8_t* device_cbc_iv;
    uint8_t* device_key_to_hack;
    uint8_t* device_return_key;
    
    printf("------------------------------------------------------- Memory allocation on device --------------------------------------------------\n");
    
    hipError_t cudaerr;

    printf("Allocation of the space for the ciphertext on the device:\t");
    // allocate device memory
    cudaerr = hipMalloc((void**)&device_ciphertext, sizeof(uint8_t) * CIPHERTEXT_LENGTH);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Allocation of the space for the plaintext on the device:\t");
    cudaerr = hipMalloc((void**)&device_plaintext, sizeof(uint8_t) * CIPHERTEXT_LENGTH);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Allocation of the space for the IVs on the device:\t\t");
    cudaerr = hipMalloc((void**)&device_cbc_iv, sizeof(uint8_t) * CIPHERTEXT_LENGTH);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Allocation of the space for the key to hack on the device:\t");
    cudaerr = hipMalloc((void**)&device_key_to_hack, sizeof(uint8_t) * AES_KEY_BYTES_LENGTH);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Allocation of the space for the key hacked on the device:\t");
    cudaerr = hipMalloc((void**)&device_return_key, sizeof(uint8_t) * AES_KEY_BYTES_LENGTH);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }

    printf("OK => Completed!\n");
    printf("--------------------------------------------------------------------------------------------------------------------------------------\n");

    printf("------------------------------------------------------ Copying data on device --------------------------------------------------------\n");

    printf("Copy of the ciphertext on the device:\t");
    cudaerr = hipMemcpy(device_ciphertext, ciphertext, sizeof(uint8_t) * CIPHERTEXT_LENGTH, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Copy of the plaintext on the device:\t");
    cudaerr = hipMemcpy(device_plaintext, plaintext, sizeof(uint8_t) * CIPHERTEXT_LENGTH, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Copy of the IVs on the device:\t\t");
    cudaerr = hipMemcpy(device_cbc_iv, ciphertext, sizeof(uint8_t) * CIPHERTEXT_LENGTH, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    printf("Copy of the key_to_hack on the device:\t");
    cudaerr = hipMemcpy(device_key_to_hack, key_to_hack, sizeof(uint8_t) * AES_KEY_BYTES_LENGTH, hipMemcpyHostToDevice);
    if (cudaerr != hipSuccess) {
        printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    else{
        printf("OK\n");
    }
    
    printf("OK => Completed!\n");
    printf("--------------------------------------------------------------------------------------------------------------------------------------\n");


    /********************************************* LAUNCH OF THE KERNEL ***********************************************/

    printf("-------------------------------------------------- Start of the brute force attack ---------------------------------------------------\n");


    // compute the maximum number of iteration in order to discover the key
    uint64_t iter_num = pow(2,NUMBER_BITS_TO_HACK);
    // maxThreadsPerBlock is the maximum number of threads per block for the current gpu
    size_t thread_per_block = (size_t)prop.maxThreadsPerBlock / 2;
    // compute the number of block to initialize
    size_t num_block = iter_num / thread_per_block;
    
    // qui si sta trovando il numero dei blocchi ma non so bene che sta facendo 
    //size_t device_setted_block_number = (message_num_block + thread_per_block - 1) / thread_per_block;

    printf("Number of block : %lu and Number of threads: %lu\n", num_block, thread_per_block);

    kernel_hack <<<num_block, thread_per_block >>> (device_ciphertext, device_plaintext, device_cbc_iv, iter_num, device_key_to_hack, device_return_key);

    hipDeviceSynchronize();

    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess){
       printf("kernel launch failed with error \"%s\".\n",hipGetErrorString(cudaerr));
    }

    //Copy the hacked key from device to host
    cudaerr = hipMemcpy(key_to_hack, device_key_to_hack, sizeof(uint8_t) * AES_KEY_BYTES_LENGTH, hipMemcpyDeviceToHost);
    if (cudaerr != hipSuccess) {
        printf("CudaMemcpy error \"%s\".\n", hipGetErrorString(cudaerr));
    }
    
    printf("--------------------------------------------------------------------------------------------------------------------------------------\n");

    /**************************************** RELEASE OF THE DEVICE ALLOCATION ****************************************/

    // release device memory
    hipFree(device_ciphertext);
    hipFree(device_plaintext);
    hipFree(device_cbc_iv);
    hipFree(device_key_to_hack);
    hipFree(device_return_key);

}